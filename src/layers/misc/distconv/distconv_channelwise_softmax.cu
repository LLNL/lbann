
#include <hip/hip_runtime.h>
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2014-2022, Lawrence Livermore National Security, LLC.
// Produced at the Lawrence Livermore National Laboratory.
// Written by the LBANN Research Team (B. Van Essen, et al.) listed in
// the CONTRIBUTORS file. <lbann-dev@llnl.gov>
//
// LLNL-CODE-697807.
// All rights reserved.
//
// This file is part of LBANN: Livermore Big Artificial Neural Network
// Toolkit. For details, see http://software.llnl.gov/LBANN or
// https://github.com/LLNL/LBANN.
//
// Licensed under the Apache License, Version 2.0 (the "Licensee"); you
// may not use this file except in compliance with the License.  You may
// obtain a copy of the License at:
//
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or
// implied. See the License for the specific language governing
// permissions and limitations under the license.
////////////////////////////////////////////////////////////////////////////////

#define LBANN_LAYERS_MISC_CHANNELWISE_SOFTMAX_INSTANTIATE

#ifdef LBANN_HAS_DISTCONV
namespace distconv{
namespace{

using Size3 = gpu_lib::array<size_t,3>;

/** @brief Max functor */
template <class T>
struct max_op {
  __device__ __forceinline__
  DataType operator()(const T& x1, const T& x2) const {
    return gpu_lib::max(x1, x2);
  }
};

} // namespace <anon>

// =========================================================
// Forward prop
// =========================================================

namespace {

/** @brief Max reduction over last dimension of 3D tensor.
 *
 *  Each CUDA block computes the max over a subset of tensor entries
 *  in @c vals and outputs the result to @c maxvals. This should be
 *  repeated multiple times to fully reduce the last tensor dimension.
 *
 *  Block dimensions: bdimx x 1 x 1
 *
 *  Grid dimensions: (vals_dims[2] / bdimx) x vals_dims[1] x vals_dims[0]
 *
 *  maxvals: vals_dims[0] x vals_dims[1] x (vals_dims[2] / bdimx)
 */
template <typename TensorDataType, size_t bdimx>
__global__ void fp_max_kernel(
  Size3 vals_dims,
  const TensorDataType* __restrict__ vals_buffer,
  Size3 vals_strides,
  TensorDataType* __restrict__ maxvals_buffer,
  Size3 maxvals_strides) {

  // Indices and dimensions
  constexpr size_t bdimy = 1;
  constexpr size_t bdimz = 1;
  const size_t tid = threadIdx.x;
  const size_t bidx = blockIdx.x;
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t gidy = threadIdx.y + blockIdx.y * blockDim.y;
  const size_t gidz = threadIdx.z + blockIdx.z * blockDim.z;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nthreadsy = blockDim.y * gridDim.y;
  const size_t nthreadsz = blockDim.z * gridDim.z;

  for (size_t k = gidz; k < vals_dims[0]; k += nthreadsz) {
    for (size_t j = gidy; j < vals_dims[1]; j += nthreadsy) {

      // Find largest value for each thread
      TensorDataType maxval{-gpu_lib::infinity<TensorDataType>()};
      for (size_t i = gidx; i < vals_dims[2]; i += nthreadsx) {
        const auto& val = vals_buffer[k * vals_strides[0]
                                      + j * vals_strides[1]
                                      + i * vals_strides[2]];
        maxval = gpu_lib::max(maxval, val);
      }

      // Find largest value for each block
      maxval = gpu_lib::block_reduce<bdimx,bdimy,bdimz,TensorDataType,max_op<TensorDataType>>(maxval);
      if (tid == 0) {
        const auto& pos = (k * maxvals_strides[0]
                           + j * maxvals_strides[1]
                           + bidx * maxvals_strides[2]);
        maxvals_buffer[pos] = maxval;
      }

    }
  }

}

/** Compute softmax.
 *
 *  y_i = exp(x_i-shift) / denom
 *
 *  Block dimensions: bdimx x bdimy x bdimz
 *
 *  Grid dimensions: (input_dims[2] / bdimx) x (input_dims[1] / bdimy) x (input_dims[0] / bdimz)
 *
 *  shifts and denoms are fully-packed 2D tensors with dimensions of
 *  input_dims[0] x input_dims[1].
 */
template <typename TensorDataType>
__global__ void fp_output_kernel(
  Size3 input_dims,
  const TensorDataType* __restrict__ input_buffer,
  Size3 input_strides,
  TensorDataType* __restrict__ output_buffer,
  Size3 output_strides,
  const TensorDataType* __restrict__ shifts,
  const TensorDataType* __restrict__ denoms) {

  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t gidy = threadIdx.y + blockIdx.y * blockDim.y;
  const size_t gidz = threadIdx.z + blockIdx.z * blockDim.z;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nthreadsy = blockDim.y * gridDim.y;
  const size_t nthreadsz = blockDim.z * gridDim.z;
  for (size_t k = gidz; k < input_dims[0]; k += nthreadsz) {
    for (size_t j = gidy; j < input_dims[1]; j += nthreadsy) {
      const auto& shift = shifts[j + k*input_dims[1]];
      const auto& denom = denoms[j + k*input_dims[1]];
      for (size_t i = gidx; i < input_dims[2]; i += nthreadsx) {
        const auto& x = input_buffer[k * input_strides[0]
                                     + j * input_strides[1]
                                     + i * input_strides[2]];
        auto& y = output_buffer[k * output_strides[0]
                                + j * output_strides[1]
                                + i * output_strides[2]];
        y = gpu_lib::exp(x-shift) / denom;
      }
    }
  }
}

} // Namespace <anon>



// =========================================================
// Backprop
// =========================================================

namespace {
/** Compute dot product between output and gradient w.r.t. output.
 *
 *  Block dimensions: bdimx x 1 x 1
 *
 *  Grid dimensions: (output_dims[2] / bdimx) x output_dims[1] x output_dims[0]
 *
 *  y_dot_dy is a fully-packed 2D tensor with dimensions of
 *  output_dims[0] x output_dims[1].
 */
template <typename TensorDataType, size_t bdimx>
__global__ void bp_y_dot_dy_kernel(
  Size3 output_dims,
  const TensorDataType* __restrict__ output_buffer,
  Size3 output_strides,
  const TensorDataType* __restrict__ output_grad_buffer,
  Size3 output_grad_strides,
  TensorDataType* __restrict__ y_dot_dy) {

  // Indices and dimensions
  constexpr size_t bdimy = 1;
  constexpr size_t bdimz = 1;
  const size_t tid = threadIdx.x;
  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t gidy = threadIdx.y + blockIdx.y * blockDim.y;
  const size_t gidz = threadIdx.z + blockIdx.z * blockDim.z;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nthreadsy = blockDim.y * gridDim.y;
  const size_t nthreadsz = blockDim.z * gridDim.z;

  for (size_t k = gidz; k < output_dims[0]; k += nthreadsz) {
    for (size_t j = gidy; j < output_dims[1]; j += nthreadsy) {

      // Compute contribution from each thread
      TensorDataType _y_dot_dy{0.};
      for (size_t i = gidx; i < output_dims[2]; i += nthreadsx) {
        const auto& y = output_buffer[k * output_strides[0]
                                      + j * output_strides[1]
                                      + i * output_strides[2]];
        const auto& dy = output_grad_buffer[k * output_grad_strides[0]
                                            + j * output_grad_strides[1]
                                            + i * output_grad_strides[2]];
        _y_dot_dy += y * dy;
      }

      // Compute contribution from each block
      _y_dot_dy = gpu_lib::block_reduce<bdimx,bdimy,bdimz>(_y_dot_dy);
      if (tid == 0) {
        gpu_lib::atomic_add(&y_dot_dy[j+k*output_dims[1]], _y_dot_dy);
      }

    }
  }

}

/** Compute gradient w.r.t. input.
 *
 *  dL/dx_i = y_i * ( dL/dy_i - dot(y,dL/dy) )
 *
 *  Block dimensions: bdimx x bdimy x bdimz
 *
 *  Grid dimensions: (output_dims[2] / bdimx) x (output_dims[1] / bdimy) x (output_dims[0] / bdimz)
 *
 *  y_dot_dy is a fully-packed 2D tensor with dimensions of
 *  output_dims[0] x output_dims[1].
 */
template <typename TensorDataType>
__global__ void bp_input_grad_kernel(
  Size3 output_dims,
  const TensorDataType* __restrict__ output_buffer,
  Size3 output_strides,
  const TensorDataType* __restrict__ output_grad_buffer,
  Size3 output_grad_strides,
  TensorDataType* __restrict__ input_grad_buffer,
  Size3 input_grad_strides,
  const TensorDataType* __restrict__ y_dot_dy) {

  const size_t gidx = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t gidy = threadIdx.y + blockIdx.y * blockDim.y;
  const size_t gidz = threadIdx.z + blockIdx.z * blockDim.z;
  const size_t nthreadsx = blockDim.x * gridDim.x;
  const size_t nthreadsy = blockDim.y * gridDim.y;
  const size_t nthreadsz = blockDim.z * gridDim.z;
  for (size_t k = gidz; k < output_dims[0]; k += nthreadsz) {
    for (size_t j = gidy; j < output_dims[1]; j += nthreadsy) {
      const auto& _y_dot_dy = y_dot_dy[j + k*output_dims[1]];
      for (size_t i = gidx; i < output_dims[2]; i += nthreadsx) {
        const auto& y = output_buffer[k * output_strides[0]
                                      + j * output_strides[1]
                                      + i * output_strides[2]];
        const auto& dy = output_grad_buffer[k * output_grad_strides[0]
                                            + j * output_grad_strides[1]
                                            + i * output_grad_strides[2]];
        auto& dx = input_grad_buffer[k * input_grad_strides[0]
                                    + j * input_grad_strides[1]
                                    + i * input_grad_strides[2]];
        dx = y * (dy - _y_dot_dy);
      }
    }
  }

}

}  // namespace <anon>

  
  template<typename Backend, typename DataType>
  template<typename Allocator>
  int
  ChannelwiseSoftmax<Backend, DataType>
  ::forward(const tensor::Tensor<DataType, tensor::LocaleMPI, Allocator> &input_0,
            tensor::Tensor<DataType, tensor::LocaleMPI, Allocator> &output){

    if (input_0.get_local_size() == 0 || output.get_local_size()){
      return 1; // no op for empty inputs
    }

    const auto& input_0_dims = input_0.get_local_shape();
    
    const auto num_channels = input_0_dims[2];
    const auto local_mini_batch_size = input_0_dims[3];
    const auto mat_channel_size = input_0_dims[0] * input_0_dims[1];
    const auto mat_stride = num_channels * mat_channel_size;

    // Convert to Hydrogen matrices for kernel launch

    using LocalMat = El::Matrix<DataType, El::Device::GPU>;

    LocalMat local_input(mat_stride,
                        local_mini_batch_size,
                        input_0.get_buffer(),
                        mat_stride);

    LocalMat local_output(mat_stride,
                          local_mini_batch_size,
                          output.get_buffer(),
                          mat_stride);
    {
      using namespace hydrogen;
      using Size3 = gpu_lib::array<size_t,3>;

      auto  multisync = MakeMultiSync(El::SyncInfoFromMatrix(local_input),
                                      El::SyncInfoFromMatrix(local_output));

      LocalMat local_shifts;
      if (!local_input.IsEmpty()) {
        constexpr size_t block_size = 256;
        dim3 block_dims, grid_dims;
        block_dims.x = block_size;
        grid_dims.x = (channel_size + block_size - 1) / block_size;
        grid_dims.y = num_channels;
        grid_dims.z = local_mini_batch_size;
        gpu_lib::clip_grid_dims(grid_dims);
        LocalMat maxvals(grid_dims.x * num_channels, local_mini_batch_size);
        hydrogen::gpu::LaunchKernel(
          fp_max_kernel<TensorDataType, block_size>,
          grid_dims, block_dims, 0, multisync,
          Size3{local_mini_batch_size, num_channels, channel_size},
          local_input.LockedBuffer(),
          Size3{static_cast<size_t>(local_input.LDim()), channel_size, 1},
          maxvals.Buffer(),
          Size3{static_cast<size_t>(maxvals.LDim()), grid_dims.x, 1});
        while (grid_dims.x > 1) {
          const size_t prev_dim = grid_dims.x;
          grid_dims.x = (prev_dim + block_size - 1) / block_size;
          const LocalMat prev_maxvals(std::move(maxvals));
          maxvals.Resize(grid_dims.x * num_channels, local_mini_batch_size);
          hydrogen::gpu::LaunchKernel(
            fp_max_kernel<TensorDataType, block_size>,
            grid_dims, block_dims, 0, multisync,
            Size3{local_mini_batch_size, num_channels, prev_dim},
            prev_maxvals.LockedBuffer(),
            Size3{static_cast<size_t>(prev_maxvals.LDim()), prev_dim, 1},
            maxvals.Buffer(),
            Size3{static_cast<size_t>(maxvals.LDim()), grid_dims.x, 1});
        }
        local_shifts = std::move(maxvals);
      }

      // Compute softmax denominators
      LocalMat local_denoms(num_channels, local_mini_batch_size);
      El::Zero(local_denoms);
      if (!local_input.IsEmpty()) {
        constexpr size_t block_size = 256;
        dim3 block_dims, grid_dims;
        block_dims.x = block_size;
        grid_dims.x = (channel_size + block_size - 1) / block_size;
        grid_dims.y = num_channels;
        grid_dims.z = local_mini_batch_size;
        gpu_lib::clip_grid_dims(grid_dims);
        hydrogen::gpu::LaunchKernel(
          fp_denom_kernel<TensorDataType, block_size>,
          grid_dims, block_dims, 0, multisync,
          Size3{local_mini_batch_size, num_channels, channel_size},
          local_input.LockedBuffer(),
          Size3{static_cast<size_t>(local_input.LDim()), channel_size, 1},
          local_shifts.LockedBuffer(),
          local_denoms.Buffer());
      }

      // Compute softmax
      if (!local_input.IsEmpty()) {
        constexpr size_t block_size = 256;
        dim3 block_dims, grid_dims;
        block_dims.x = block_size;
        grid_dims.x = (channel_size + block_size - 1) / block_size;
        grid_dims.y = num_channels;
        grid_dims.z = local_mini_batch_size;
        gpu_lib::clip_grid_dims(grid_dims);
        hydrogen::gpu::LaunchKernel(
          fp_output_kernel<TensorDataType>,
          grid_dims, block_dims, 0, multisync,
          Size3{local_mini_batch_size, num_channels, channel_size},
          local_input.LockedBuffer(),
          Size3{static_cast<size_t>(local_input.LDim()), channel_size, 1},
          local_output.Buffer(),
          Size3{static_cast<size_t>(local_output.LDim()), channel_size, 1},
          local_shifts.LockedBuffer(),
          local_denoms.LockedBuffer());
      }
 
    }  // namespace hydrogen
    return 1;        
  }

  template<typename Backend, typename DataType>
  template<typename Allocator>
  int
  ChannelwiseSoftmax<Backend, DataType>
  ::backward(const tensor::Tensor<DataType, tensor::LocaleMPI, Allocator> &input_0,
             const tensor::Tensor<DataType, tensor::LocaleMPI, Allocator> &output_grad,
             tensor::Tensor<DataType, tensor::LocaleMPI, Allocator> &input_grad_0){

    return 1;        
  }

// =========================================================
// Explicit template instantiation
// =========================================================
}  // namespace distconv
#endif // LBANN_HAS_DISTCONV